
#include <hip/hip_runtime.h>
#include <iostream>

/**
 * @brief Perform general 1-D grid, 2-D block reduce, along X-direction.
 *
 * @details This device function implements the reduce algorithms. The grid is
 * in 1-D X-direction, i.e. `gridDim.x >= 1`,`gridDim.y == 1` and
 * `gridDim.z == 1`. The block is in 2-D X- and Y-direction, i.e.
 * `blockDim.x >= 1` and `blockDim.y >= 1`, `blockDim.z == 1`. To achieve higher
 * parallelization, the shared memory will have size
 * `blockDim.x * blockDim.y * 2`. The input array will be *divided* into smaller
 * parts and assigned to each thread block. That is, each block will be assigned
 * an *sub-array* with length at most `blockDim.x * blockDim.y * 2`.
 *
 * @tparam      DataType        The type of data, which is processed.
 * @tparam      Operation       The operation type, it is related to the lambda
 * function parameter.
 *
 * @param[in]   d_inputArr      The input array, type is determined by the
 * template.
 *
 * @param[in]   inputArrLength  The length of the input array.
 * @param[out]  d_outputArr     The output array, in device memory, type is
 * determined by the template. The length of it must be `gridDim.x * blockDim.y`
 *
 * @param[in]   oper            The operation performed on two elements in the
 * array. It should be a function or lambda expression defined in a
 * `__global__` function, which has two reference parameters. The result on
 * these two parameters will be stored in the first parameter, the second
 * parameter should be set to identity.
 *
 * @param[in]   identity        The identity of the operation. (The same concept
 * in group theory.)
 *
 */
template < class DataType, class Operation >
__device__
void
g1b2_reduce_x(
        const DataType *const   d_inputArr,
        const size_t            inputArrLength,
        DataType *const         d_outputArr,
        const Operation         &oper,
        const DataType          identity
        ) {
    // Make shared memory visible in kernel
    extern __shared__ DataType sdata[];

    // Calculate indices
    int sdataAbsIdx = threadIdx.x * blockDim.y + threadIdx.y;
    int initStride = blockDim.x * blockDim.y;
    int inputArrAbsIdx = 2 * blockIdx.x * initStride + sdataAbsIdx;

    // Initialize shared memory according to the absolute location of
    // thread-releated element in the input array. If a thread-related element
    // does not exist, the corresponding shared memory will be filled with
    // identity value of the operation.
    if (inputArrAbsIdx >= inputArrLength) {
        // The thread-related left element does not exist in the input array.
        sdata[sdataAbsIdx] = identity;
        sdata[sdataAbsIdx + initStride] = identity;
    }
    else if (inputArrAbsIdx + initStride >= inputArrLength) {
        // The thread-related right element does not exist in the input array.
        sdata[sdataAbsIdx] = d_inputArr[inputArrAbsIdx];
        sdata[sdataAbsIdx + initStride] = identity;
    }
    else {
        // The thread-related elements exist in the input array.
        sdata[sdataAbsIdx] = d_inputArr[inputArrAbsIdx];
        sdata[sdataAbsIdx + initStride]
            = d_inputArr[inputArrAbsIdx + initStride];
    }
    // Thread synchronization, wait for shared memory initialization's finish.
    __syncthreads();

    // Perform reduce, the initial stride is blockDim.x
    for (unsigned int stride = static_cast<unsigned int>(blockDim.x);
        stride > 0;
        stride = (stride & 0x1) ? stride >> 1 + 1 : stride >> 1) {
        // The thread with X-ID smaller than the stride will be omitted.
        if (threadIdx.x < stride)
            // Compute absolute stride and perform operation
            oper(sdata[sdataAbsIdx], sdata[sdataAbsIdx + stride * blockDim.y]);
        // Thread synchronization after every stride.
        __syncthreads();
    }

    // Because reduce is executed in X-direction, only the elements in shared
    // memory related to the `threadIdx.x == 0` stored the results. Here only
    // the corresponding elements will be written into the output array.
    if (0 == threadIdx.x)
        d_outputArr[blockIdx.x * blockDim.y + threadIdx.y] = sdata[sdataAbsIdx];
}

#define HIST_WIDTH      128
#define HIST_NUM        8
#define BLOCK_NUM       512
#define DATA_LENGTH     (2 * HIST_WIDTH * HIST_NUM * BLOCK_NUM + HIST_WIDTH * 3)
#define RESULT_LENGTH   HIST_WIDTH * (BLOCK_NUM + 1)

__global__
void
add_oper(
        const int *const        d_inputArr,
        const size_t            inputArrLength,
        int *const              d_outputArr
        ) {
    // Identity and lambda of the operation.
    const int identity = 0;
    auto oper = [](int &l, int &r) -> void { l += r; r = 0; };

    // Launch the device function.
    g1b2_reduce_x(d_inputArr, DATA_LENGTH, d_outputArr, oper, identity);
}

int main() {
    // All kinds of sizes.
    // Shared memory's size (in bytes).
    int sharedBytes = 2 * sizeof(int) * HIST_WIDTH * HIST_NUM;

    // Input array's size (in bytes).
    int dataBytes   = sizeof(int) * DATA_LENGTH;

    // Result array's size (in bytes).
    int resultBytes = sizeof(int) * RESULT_LENGTH;

    // Data arrays
    // Input array, host.
    int h_inputArr[DATA_LENGTH];
    // Output array, host.
    int h_outputArr[RESULT_LENGTH];

    // Initialization of input array.
    for (int idx = 0; idx < DATA_LENGTH; ++idx)
        h_inputArr[idx] = idx % HIST_WIDTH;


    // CUDA kernel sizes
    // Grid size
    const dim3 gridSize(BLOCK_NUM);
    // Thread block size
    const dim3 blockSize(HIST_NUM, HIST_WIDTH);

    // Allocating device memory
    int *d_inputArr, *d_outputArr;
    int ret = hipMalloc(&d_inputArr, dataBytes);
    if (hipSuccess != ret)
        std::cout << "Cannot allocate d_inputArr, ret: " << ret << std::endl;
    ret = hipMalloc(&d_outputArr, resultBytes);
    if (hipSuccess != ret)
        std::cout << "Cannot allocate d_outputArr, ret: " << ret << std::endl;

    // Copy host array to device
    ret = hipMemcpy(d_inputArr, h_inputArr, dataBytes, hipMemcpyHostToDevice);
    if (hipSuccess != ret)
        std::cout << "Cannot memcpy to device, ret: " << ret << std::endl;

    add_oper<<< BLOCK_NUM + 1, blockSize, sharedBytes >>>(
            d_inputArr, DATA_LENGTH, d_outputArr);

    ret = hipMemcpy(h_outputArr, d_outputArr, resultBytes,
            hipMemcpyDeviceToHost);
    if (hipSuccess != ret)
        std::cout << "Cannot memcpy to host, ret: " << ret << std::endl;

    for (int idx = RESULT_LENGTH - 512; idx < RESULT_LENGTH; ++idx)
        std::cout << h_outputArr[idx] << " ";
    std::cout << std::endl;

    hipFree(d_inputArr);
    hipFree(d_outputArr);

    return 0;
}
